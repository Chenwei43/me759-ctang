#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

#include "hip/hip_runtime.h"
#include ""
#include "reduce.cuh"

int main(int argc, char* argv[]) {
    unsigned int n;
    unsigned int threads_per_block;
    std::istringstream input_1(argv[1]);
    std::istringstream input_2(argv[2]);
    if (input_1 >> n && input_2>>threads_per_block && input_2.eof()) {
        //create arrays on host, pinned mem
        float *inputHost = new float[n];
        //hipHostMalloc(&inputHost, sizeof(float) * n);

        std::random_device entropy_source;
        std::mt19937 generator(entropy_source());
        std::uniform_real_distribution<float> dist(-1.0, 1.0);
        for (unsigned int i = 0; i < n; i++) {
            inputHost[i] = dist(generator);
            //inputHost[i] = 1.0;
        }

        // cuda arrays
        unsigned int numBlocks = 1;
        if (threads_per_block < n) {
            numBlocks = (n + threads_per_block - 1) / (2 * threads_per_block);
        }
        float *input, *output;
        hipMalloc((void**)&input, sizeof(float ) * n);
        hipMalloc((void**)&output, sizeof(float ) * numBlocks);	
        hipMemcpy(input, inputHost, sizeof(float)*n, hipMemcpyHostToDevice);
        hipMemset(output, 0, sizeof(int) * numBlocks);
        /*float* test = new float[n];
        hipMemcpy(test, input, sizeof(float*) * n, hipMemcpyDeviceToHost);
        std::cout << test[0] << "\n";*/
        
        
        // kernel call
        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        reduce(&input, &output, n, threads_per_block);

        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
        hipEventDestroy(startEvent);
        hipEventDestroy(stopEvent);

        float result;
        hipMemcpy(&result, input, sizeof(float), hipMemcpyDeviceToHost);
        std::cout << result << "\n";
        std::cout << elapsedTime << "\n";
        
        //delete[] test;

        hipFree(output);
        hipFree(input);
        
    }
    return 0;
}