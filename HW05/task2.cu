#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

#include "hip/hip_runtime.h"
#include ""
#include "matmul.cuh"

int main(int argc, char* argv[])
{
    unsigned int n;
    unsigned int block_dim;
    std::istringstream input_1(argv[1]);
    std::istringstream input_2(argv[2]);
    if (input_1 >> n && input_2>>block_dim && input_2.eof()) {
                //create arrays on managed mem
        int* A, * B, * C;
        hipMallocManaged(&A, sizeof(int) * n * n);
        hipMallocManaged(&B, sizeof(int) * n * n);
        hipMallocManaged(&C, sizeof(int) * n * n);

        std::random_device entropy_source;
        std::mt19937 generator(entropy_source());
        std::uniform_int_distribution<int> dist1(-10,10);
        for (unsigned int i = 0; i < n*n; i++) {
            A[i] = dist1(generator);
            B[i] = dist1(generator);

        }

        // kernel call
        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);

        matmul_1(A, B, C, n, block_dim);

        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
        hipEventDestroy(startEvent);
        hipEventDestroy(stopEvent);

        std::cout << C[0] << "\n";
        std::cout << C[n * n - 1] << "\n";
        std::cout << elapsedTime << "\n";

        hipFree(A);
        hipFree(B);
        hipFree(C);
        
        //Float
        float *Af, *Bf, *Cf;
        hipMallocManaged(&Af, sizeof(float) * n * n);
        hipMallocManaged(&Bf, sizeof(float) * n * n);
        hipMallocManaged(&Cf, sizeof(float) * n * n);

        std::uniform_real_distribution<float> dist2(-1.0, 1.0);
        for (unsigned int i = 0; i < n * n; i++) {
            Af[i] = dist2(generator);
            Bf[i] = dist2(generator);
        }

        hipEvent_t startEvent2, stopEvent2;
        hipEventCreate(&startEvent2);
        hipEventCreate(&stopEvent2);
        hipEventRecord(startEvent2, 0);

        matmul_2(Af, Bf, Cf, n, block_dim);

        hipEventRecord(stopEvent2, 0);
        hipEventSynchronize(stopEvent2);
        float elapsedTime2;
        hipEventElapsedTime(&elapsedTime2, startEvent2, stopEvent2);
        hipEventDestroy(startEvent2);
        hipEventDestroy(stopEvent2);

        std::cout << Cf[0] << "\n";
        std::cout << Cf[n * n - 1] << "\n";
        std::cout << elapsedTime2 << "\n";

        hipFree(Af);
        hipFree(Bf);
        hipFree(Cf);

        //Double
        double *Ad, *Bd, *Cd;
        hipMallocManaged(&Ad, sizeof(double) * n * n);
        hipMallocManaged(&Bd, sizeof(double) * n * n);
        hipMallocManaged(&Cd, sizeof(double) * n * n);

        std::uniform_real_distribution<double> dist3(-1.0, 1.0);
        for (unsigned int i = 0; i < n * n; i++) {
            Ad[i] = dist3(generator);
            Bd[i] = dist3(generator);
        }

        // kernel call
        hipEvent_t startEvent3, stopEvent3;
        hipEventCreate(&startEvent3);
        hipEventCreate(&stopEvent3);
        hipEventRecord(startEvent3, 0);

        matmul_3(Ad, Bd, Cd, n, block_dim);

        hipEventRecord(stopEvent3, 0);
        hipEventSynchronize(stopEvent3);
        float elapsedTime3;
        hipEventElapsedTime(&elapsedTime3, startEvent3, stopEvent3);
        hipEventDestroy(startEvent3);
        hipEventDestroy(stopEvent3);

        std::cout << Cd[0] << "\n";
        std::cout << Cd[n * n - 1] << "\n";
        std::cout << elapsedTime3 << "\n";

        hipFree(Ad);
        hipFree(Bd);
        hipFree(Cd);
        
    }

    return 0;
}