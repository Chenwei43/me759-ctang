#include "hip/hip_runtime.h"
#include <vector>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/random/linear_congruential_engine.h>

int main()
{   
    //const unsigned int nproj = 9990;
    const float B0 = .3;
    const unsigned int Npe = 110;
    const unsigned int Nfreq = 110;
    const unsigned int Npts = Npe * Nfreq;
    const unsigned int threads_per_block = 1024;
    const unsigned int res = 256;
    const float fr = 0.9 * Nfreq / 2.0;
    const float fw = 0.1 * Nfreq / 2.0;

    // kspace coords - TODO switch to vds
    const float Tread = .3;     // in sec
    float* kx, * ky;
    hipMallocManaged(&kx, sizeof(float) * Npts);
    hipMallocManaged(&ky, sizeof(float) * Npts);
    float tres = Tread / (Npts);
    for (unsigned int i = 0; i < Npts; i++) {
        float tt = sqrt((0.0 + tres * i) / Tread);
        kx[i] = Nfreq / 2 * tt * cos(2 * PI * Nfreq / 2 * tt);
        ky[i] = - Nfreq / 2 * tt * sin(2 * PI * Nfreq / 2 * tt);
    }

    // gradient waveform
    float* gx, * gy;
    hipMallocManaged(&gx, sizeof(float) * Npts);
    hipMallocManaged(&gy, sizeof(float) * Npts);
    for (unsigned int i = 0; i < Npts; i++) {
        if (i != Npts - 1) {
            gx[i] = 1 / GAMMA_BAR * (kx[i] - kx[i + 1]) / tres;
            gy[i] = 1 / GAMMA_BAR * (ky[i] - ky[i + 1]) / tres;
        }
        else {
            gx[i] = 1 / GAMMA_BAR * (kx[i] - 0.f) / tres;
            gy[i] = 1 / GAMMA_BAR * (ky[i] - 0.f) / tres;
        }        
    }
    

    // dcf 
    float* wx, * wy;
    hipMallocManaged(&gx, sizeof(float) * Npts);
    hipMallocManaged(&gy, sizeof(float) * Npts);

    unsigned int numBlocks = (Npts + threads_per_block - 1) / threads_per_block;
    get_dcf<<<numBlocks, threads_per_block, 2 * threads_per_block * sizeof(float) >>> (kx, ky, fr, fw, wx, wy);
    hipDeviceSynchronize();


    //logical coords
    float* x, * y, * z;
    hipMallocManaged(&x, sizeof(float) * res * res);
    hipMallocManaged(&y, sizeof(float) * res * res);
    hipMallocManaged(&z, sizeof(float) * res * res);    
    for (unsigned int i = 0; i < res * res; i++) {
        x[i] = -0.5 + 1 / res * (i % res);
        y[i] = -0.5 + 1 / res * (i / res);
        z[i] = 1.0;

    }

    //transform matrix to physical coords
    float affine[9] = {.3, 0, 0, 0, .3, 0, 0, 0, .3};
    float* affine_cu;
    hipMalloc((void**)&affine_cu, sizeof(float) * 9);
    hipMemcpy(affine_cu, &affine, sizeof(float) * 9, hipMemcpyHostToDevice);

    // Phi_c(xres, yres, t)
    float* Phi_c;
    hipMallocManaged(&Phi_c, sizeof(float) * res * res * Npts);
    getBc_spiral(gx, gy, x, y, z, tres, B0, affine, Phi_c);

    for (unsigned int i = 0; i < Npts; i++) {

        // acquisition


        // recon


    }
    


    return 0;
}

