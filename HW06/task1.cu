// matmul by cuBLAS
#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include "mmul.h"

int main(int argc, char* argv[]) {
    unsigned int n = std::strtoul(argv[1], nullptr, 10);
    unsigned int n_tests = std::atoi(argv[2]);
    
    float* A, * B, * C;
    hipMallocManaged(&A, sizeof(float) * n * n);
    hipMallocManaged(&B, sizeof(float) * n * n);
    hipMallocManaged(&C, sizeof(float) * n * n);

    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);

    //cuBLAS is col major order. ii is col index, although it doesn't really matter here
    for (unsigned int ii = 0; ii < n ; ii++) {
        for (unsigned int jj = 0; jj < n; jj++) {
            A[ii * n + jj] = dist(generator);
            B[ii * n + jj] = dist(generator);
            C[ii * n + jj] = dist(generator);
        }
    }

    float running_time = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (unsigned int trial = 0; trial < n_tests; trial++) {
        hipEvent_t startEvent, stopEvent;
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
        hipEventRecord(startEvent, 0);
        
        mmul(handle, A, B, C, n);

        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
        hipEventDestroy(startEvent);
        hipEventDestroy(stopEvent);

        running_time += elapsedTime;
    }
    running_time /= n_tests;
    std::cout << running_time << "\n";


    hipblasDestroy(handle);
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}