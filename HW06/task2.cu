#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

#include "hip/hip_runtime.h"
#include ""
#include "scan.cuh"

int main(int argc, char* argv[]) {
    unsigned int n = std::strtoul(argv[1], nullptr, 10);
    unsigned int threads_per_block = std::atoi(argv[2]);

    float* input, * output;
    hipMallocManaged(&input, sizeof(float) * n);
    hipMallocManaged(&output, sizeof(float) * n);

    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);
    for (unsigned int ii = 0; ii < n; ii++) {
        input[ii] = dist(generator);
    }

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    scan(input, output, n, threads_per_block);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    /*
    // correctness checked 
    for (unsigned int i = 0; i < n; i++) {
        std::cout << input[i] << " " << output[i] << "\n";
    }
    */
    
    std::cout << output[n-1] << "\n";
    std::cout << elapsedTime << "\n";

    hipFree(input);
    hipFree(output);

    return 0;
}