#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

#include "hip/hip_runtime.h"
#include ""
#include "scan.cuh"


__global__ void hillis_steele_blocks(float* g_odata, const float* g_idata, float *eob) {
    /* **inclusive** scan of each block */

    extern volatile __shared__  float temp[]; // allocated on invocation

    int thid = threadIdx.x;
    int idxFull = threadIdx.x + blockIdx.x * blockDim.x;


        int pout = 0, pin = 1;

        // load input into shared memory. when overshoot, write 0s. 
        temp[thid] = g_idata[idxFull];
        __syncthreads();

        for (int offset = 1; offset < blockDim.x; offset *= 2) {
            pout = 1 - pout; // swap double buffer indices
            pin = 1 - pout;

            if (thid >= offset)
                temp[pout * blockDim.x + thid] = temp[pin * blockDim.x + thid] + temp[pin * blockDim.x + thid - offset];
            else
                temp[pout * blockDim.x + thid] = temp[pin * blockDim.x + thid];

            __syncthreads(); // I need this here before I start next iteration 
        }

        // crop the padded
            g_odata[idxFull] = temp[pout * blockDim.x + thid];
    
        __syncthreads();

        eob[blockIdx.x] = g_odata[blockDim.x * (1+blockIdx.x)-1];
    
    
}

__global__ void hillis_steele_eob(float* g_odata, const float* g_idata, unsigned int n) {
    /* **inclusive** scan of array consists of last entries of each block 
        Under the assumption that length of input <= threads_per_block**2, this can be done safely by 1 block.
    */

    extern volatile __shared__  float temp[]; // allocated on invocation

    int thid = threadIdx.x;
    int pout = 0, pin = 1;


    // load input into shared memory. 
    temp[thid] = g_idata[thid];
 
    __syncthreads();

    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        pout = 1 - pout; // swap double buffer indices
        pin = 1 - pout;

        if (thid >= offset)
            temp[pout * blockDim.x + thid] = temp[pin * blockDim.x + thid] + temp[pin * blockDim.x + thid - offset];
        else
            temp[pout * blockDim.x + thid] = temp[pin * blockDim.x + thid];

        __syncthreads(); // I need this here before I start next iteration 
    }
    
    g_odata[thid] = temp[pout * blockDim.x + thid];
}

/* add scanned eob to individually scanned blocks -> final results */
__global__ void add_eob(float* g_odata, float* eob, unsigned int threads_per_block, unsigned int numBlocks) {

    if (blockIdx.x < (numBlocks - 1)) {
        g_odata[(blockIdx.x+1)*threads_per_block+threadIdx.x] += eob[blockIdx.x];
    }    

}

/* Added last entry of scanned eob to entries of residual block*/
__global__ void add_lastFull(float* g_odata, float* eob, unsigned int num_fullBlocks) {

    g_odata[threadIdx.x] += eob[num_fullBlocks-1];    

}


__host__ void scan(const float* input, float* output, unsigned int n, unsigned int threads_per_block) {

    
    unsigned int num_entries_last_block = n % threads_per_block;
    if (num_entries_last_block == 0) {

        //scan each block
        unsigned int numBlocks = (n + threads_per_block - 1) / threads_per_block;
        
        float* end_of_block, * end_of_block_scanned;
        hipMalloc((void**)&end_of_block, numBlocks * sizeof(float));
        hipMalloc((void**)&end_of_block_scanned, numBlocks * sizeof(float));

        hillis_steele_blocks<<<numBlocks, threads_per_block, 2 * threads_per_block * sizeof(float)>>> (output, input, end_of_block);

        //scan array composed of the last entries of each block
        //under the assumption, it can be done with a single block
        // unsigned int numBlocks_scanBlock = (numBlocks + threads_per_block - 1) / (2 * threads_per_block);
        hillis_steele_eob<<<1, numBlocks, 2 * numBlocks * sizeof(float)>>>(end_of_block_scanned, end_of_block, numBlocks);


        //Add end of each block to corresponding entries
        add_eob<<<numBlocks, threads_per_block >>>(output, end_of_block_scanned, threads_per_block, numBlocks);


        hipFree(end_of_block);
        hipFree(end_of_block_scanned);
    }
    else
    {
        unsigned int real_n = n - num_entries_last_block;
        unsigned int num_fullBlocks = (real_n + threads_per_block - 1) / threads_per_block;

        // scan the first real_n full blocks
        float* end_of_block, * end_of_block_scanned;
        hipMalloc((void**)&end_of_block, num_fullBlocks * sizeof(float));
        hipMalloc((void**)&end_of_block_scanned, num_fullBlocks * sizeof(float));
        hillis_steele_blocks<<<num_fullBlocks, threads_per_block, 2 * threads_per_block * sizeof(float)>>>(output, input, end_of_block);
        hillis_steele_eob<<<1, num_fullBlocks, 2 * num_fullBlocks * sizeof(float)>>>(end_of_block_scanned, end_of_block, num_fullBlocks);
        add_eob<<<num_fullBlocks, threads_per_block>>>(output, end_of_block_scanned, threads_per_block, num_fullBlocks);

        // scan the last block
        float* last_block_scanned = &(output[real_n]);
        const float* last_block = &(input[real_n]);        
        hillis_steele_eob<<<1, num_entries_last_block, 2 * num_entries_last_block * sizeof(float)>>>(last_block_scanned, last_block, num_entries_last_block);

        // Add last entry of  end_of_block_scanned to last_block_scanned -> the residual block is scanned. Then stitch full blocks and residual block.
        add_lastFull<<<1, num_entries_last_block>>>(last_block_scanned, end_of_block_scanned, num_fullBlocks);

        hipFree(end_of_block);
        hipFree(end_of_block_scanned);

    }  
    hipDeviceSynchronize();

}