#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include "mmul.h"

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n) {

	int lda = n, ldb = n, ldc = n;
	const float alphaValue = 1.0, betaValue = 1.0;
	const float* alpha = &alphaValue;
	const float* beta = &betaValue;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A, lda, B, ldb, beta, C, ldc);

}