// Thrust reduce
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/random/linear_congruential_engine.h>

#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

__host__ static __inline__ float rand_pm1()
{
    return (float)(rand()) / (float)(RAND_MAX/2) - 1.f;
}

int main(int argc, char* argv[]) {
    // std::ofstream timefile;
    // timefile.open("timingTask1_thrust.txt");
    // for (int j = 10; j < 31; j++) {

    unsigned int n = std::strtoul(argv[1], nullptr, 10);

    thrust::host_vector<float> h_vec(n);
    thrust::generate(h_vec.begin(), h_vec.end(), rand_pm1);
    
    thrust::device_vector<float> d_vec =  h_vec;
    float init = d_vec[0];

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    float result = thrust::reduce(d_vec.begin(), d_vec.end(), init);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    // timefile << n << " " << elapsedTime << "\n";
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    std::cout << result << std::endl;
    std::cout << elapsedTime << std::endl;
    // }

    return 0;
}