// compute a * x + y
#include "hip/hip_runtime.h"



#include <iostream>
#include <random>
#include <cstdio>

__global__ void mulAdd(int* dA, int a, int Nthreads) {
    dA[blockIdx.x * Nthreads + threadIdx.x] = blockIdx.x + a*threadIdx.x;    
}

int main() {
    const int numThreads = 8;
    const int numBlocks = 2;
    const int lenArray = 16;
    int hA[lenArray];

    // cuda arrays
    int* dA;
    hipMalloc((void**)&dA, sizeof(int) * lenArray);  
    hipMemset(dA, 0, sizeof(int) * lenArray);

    int my_seed = 137;
    std::mt19937 generator(my_seed);
    const int min = -100, max = 100;
    std::uniform_int_distribution<int> dist(min, max);
    int a = dist(generator);
    
    mulAdd<<<numBlocks, numThreads>>>(dA, a, numThreads);

    hipMemcpy(&hA, dA, sizeof(int) * lenArray, hipMemcpyDeviceToHost);
    for (unsigned int i = 0; i < lenArray; i++){
        std::cout << hA[i] << " ";
    }
    std::cout << "\n";

    hipFree(dA);
    return 0;
}
